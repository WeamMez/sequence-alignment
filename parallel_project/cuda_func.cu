#include "hip/hip_runtime.h"
#include "cuda_func.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__device__ CROSS_RESULT cuda_cross(const Letter *main_letter, const Letter *other_letter)
{
    if (main_letter->letter == MUTANT_CHAR || other_letter->letter == MUTANT_CHAR)
        return Not_to_count;
    if (main_letter->letter == other_letter->letter)
        return Identical;
    int8_t i;
    for (i = CROSS_RESULT_SPECIALS; i < types_of_groups + CROSS_RESULT_SPECIALS; i++)
        if (main_letter->in_groups[i - CROSS_RESULT_SPECIALS] & (other_letter->in_groups[i - CROSS_RESULT_SPECIALS]))
            return CROSS_RESULT(i);
    return None; // according to CROSS_RESULT
}

__global__ void cross_kernel(const Letter **main_seq, int main_size, const Letter **other_seq, int other_size, int *weights,
    int offset, int mutation, int *ans)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= other_size)
        return;
    int my_weight;
    if (i >= mutation)
        my_weight = (int)weights[cuda_cross((main_seq)[i + offset + 1], (other_seq)[i])];
    else
        my_weight = (int)weights[cuda_cross((main_seq)[i + offset], (other_seq)[i])];

    atomicAdd(ans, my_weight);
}

void seq_cross(const Letter **main_seq, int main_size, const Letter **other_seq, int other_size, int *weights, int offset, int mutation, int *cuda_reduced_ans_temp)
{
    int compare_until = other_size;

    unsigned int blocks = DIV_ROUND_UP(compare_until, THREADS_PER_BLOCK);
    hipMemset(cuda_reduced_ans_temp, 0, sizeof(int));

    cross_kernel<<<blocks, THREADS_PER_BLOCK>>>(main_seq, main_size, other_seq, other_size, weights, offset, mutation, cuda_reduced_ans_temp);

    hipDeviceSynchronize();
}

__global__ void seq_to_cuda_kernel(const Letter *const *letter, int size, const Letter *cpu_alphabet_first_letter, const Letter *cuda_alphabet_first_letter, Letter **target)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
        target[i] = (Letter *)(cuda_alphabet_first_letter + (letter[i] - cpu_alphabet_first_letter)); // Calculate the address of the letter in the cuda_alphabet
}

Letter **seq_to_cuda(const Alphabet *cuda_alphabet, const Sequence *seq)
{
    Letter **cuda_seq, **cuda_host_sequence;
    hipMalloc(&cuda_seq, sizeof(Letter *) * seq->size());
    hipMalloc(&cuda_host_sequence, sizeof(Letter *) * seq->size());
    hipMemcpy(cuda_host_sequence, seq->sequence.data(), sizeof(Letter *) * seq->size(), hipMemcpyHostToDevice);

    unsigned int blocks = DIV_ROUND_UP(seq->size(), THREADS_PER_BLOCK);
    seq_to_cuda_kernel<<<blocks, THREADS_PER_BLOCK>>>(cuda_host_sequence, seq->size(), seq->alphabet->letters, cuda_alphabet->letters, cuda_seq);
    hipDeviceSynchronize();

    return cuda_seq;
}

Fit_result seq_fit(const Sequence *main_seq, const Sequence *other_seq, array<int, CROSS_RESULT_COUNT> *weights)
{

    int max_offset = 0, max_mutation = 0;
    int max_score = numeric_limits<int>::min(), score;


    Alphabet *cuda_alphabet;
    hipMalloc(&cuda_alphabet, sizeof(Alphabet));
    hipMemcpy(cuda_alphabet, main_seq->alphabet, sizeof(Alphabet), hipMemcpyHostToDevice);

    Letter **cuda_main_seq = seq_to_cuda(cuda_alphabet, main_seq);
    Letter **cuda_other_seq = seq_to_cuda(cuda_alphabet, other_seq);

    int *cuda_weights;
    hipMalloc(&cuda_weights, sizeof(int) * CROSS_RESULT_COUNT);
    hipMemcpy(cuda_weights, weights, sizeof(int) * CROSS_RESULT_COUNT, hipMemcpyHostToDevice);

    int *cuda_reduced_ans_temp;
    hipMalloc(&cuda_reduced_ans_temp, sizeof(int));

    #pragma omp parallel for
    for (int offset = 0; offset < (int)(main_seq->size()) - other_seq->size(); offset++)
        for (int mutation = 1; mutation <= (int)(other_seq->size()); mutation++)
        {
            seq_cross((const Letter **)cuda_main_seq, main_seq->size(), (const Letter **)cuda_other_seq, other_seq->size(), cuda_weights, offset, mutation, cuda_reduced_ans_temp);
            hipMemcpy(&score, cuda_reduced_ans_temp, sizeof(int), hipMemcpyDeviceToHost);
            
#ifdef LONG_DEBUG
            printf("\toffset = %zd\tmutation = %zd\tscore = %d\n", offset, mutation, score);
#endif
            #pragma omp critical
            if (score > max_score)
            {
                max_score = score;
                max_offset = offset;
                max_mutation = mutation;
            }
        }
    seq_cross((const Letter **)cuda_main_seq, main_seq->size(), (const Letter **)cuda_other_seq, other_seq->size(), cuda_weights, main_seq->size() - other_seq->size(), other_seq->size(), cuda_reduced_ans_temp);
    hipMemcpy(&score, cuda_reduced_ans_temp, sizeof(int), hipMemcpyDeviceToHost);
    if (score > max_score)
    {
        max_score = score;
        max_offset = main_seq->size() - other_seq->size();
        max_mutation = other_seq->size();
    }

    hipFree(cuda_main_seq);
    hipFree(cuda_other_seq);
    hipFree(cuda_weights);
    hipFree(cuda_alphabet);

    return {max_offset, max_mutation, max_score};
}